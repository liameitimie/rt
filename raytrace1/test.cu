#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include "cudaChecker.h"

#include <iostream>
#include "myMath.h"
#include "Ray.h"
#include "Sphere.h"
#include "Camera.h"

using namespace std;

#define WIDTH 1024
#define HEIGHT 768
#define SAMPLE_PER_PIXCEL 1024
#define PI 3.141592653589793f

Sphere* d_s;
Sphere h_s[]={
	//    �뾶  λ��                           �Է���          ��ɫ                     ���� 
	Sphere(1e5, make_float3( 1e5+1,40.8,81.6), make_float3(0), make_float3(1.0,0.35,0.35),DIFF),//��ǽ�� 
	Sphere(1e5, make_float3(-1e5+99,40.8,81.6),make_float3(0), make_float3(0.35,0.35,1.0),DIFF),//��ǽ�� 
	Sphere(1e5, make_float3(50,40.8, 1e5),     make_float3(0), make_float3(1.0,1.0,1.0),DIFF),//��ǽ�� 
	Sphere(1e5, make_float3(50,40.8,-1e5+250), make_float3(0), make_float3(0),          DIFF),//ǰǽ�� 
	Sphere(1e5, make_float3(50, 1e5, 81.6),    make_float3(0), make_float3(1.0,1.0,1.0),DIFF),//�ذ�
	Sphere(1e5, make_float3(50,-1e5+81.6,81.6),make_float3(0), make_float3(1.0,1.0,1.0),DIFF),//�컨��
	Sphere(16.5,make_float3(27,16.5,47),       make_float3(0), make_float3(1,1,1)*.999, SPEC),//������ 
	Sphere(16.5,make_float3(73,16.5,78),       make_float3(0), make_float3(1,1,1)*.999, DIFF),//�������� 
	Sphere(600, make_float3(50,681.6-.27,81.6),make_float3(12,12,12),   make_float3(0), DIFF) //��Դ
};

int n=sizeof(h_s)/sizeof(Sphere);

float4 h_c[HEIGHT][WIDTH];
float4* d_c;

float* d_samples;//������ 

hiprandStateXORWOW_t* states;//�����������״̬
__global__
void init_curand(hiprandStateXORWOW_t* states,unsigned long long seed){
	int idx=threadIdx.x;
	hiprand_init(seed,idx,0,&states[idx]);
}


__device__
bool closestHit(const Ray& r,Sphere* d_s,int n,float& t,int& id){
	bool hitted=false;
	float tmin=1e-4,tmax=1e10;
	for(int i=0;i<n;i++){
		float tmp=intersect(d_s[i],r,tmin,tmax);
		if(tmp>0){
			hitted=true,tmax=tmp;
			t=tmp,id=i;
		}
	}
	return hitted;
}

__global__
void castRay(Sphere* d_s,int n,float4* d_c,Camera cam,float* d_samples,hiprandStateXORWOW_t* states){
	int x=blockIdx.x,y=blockIdx.y;
	int idx=x+y*WIDTH;
	
	__shared__ float4 c[SAMPLE_PER_PIXCEL];
	int ti=threadIdx.x;

	int dep=0;
	float dx=d_samples[ti];
	float dy=d_samples[2*SAMPLE_PER_PIXCEL-ti-1];
	Ray r=cam.genRay(x+dx,y+dy,WIDTH,HEIGHT);
	float3 cl=make_float3(0.0f),cf=make_float3(1.0f);
	
	while(1){
		float t;int id;
		if(!closestHit(r,d_s,n,t,id)) break;
		const Sphere& obj=d_s[id];
		
		cl+=cf*obj.e;
		cf*=obj.c;
		if(++dep>6) break;
		
		float3 p=r.o+r.d*t;
		float3 tn=normalize(p-obj.p);
		float3 n=dot(tn,r.d)<0?tn:-tn;
		p+=n*make_float3(1e-1);
		
		if(obj.refl==SPEC){
			float3 d=r.d-n*(2*dot(n,r.d));
			r=Ray(p,d);
		}
		else{
			float r1=2*PI*hiprand_uniform(&states[ti]);
			float r2=hiprand_uniform(&states[ti]);
			
			float3 u=normalize(cross(n,(abs(n.x)>0.5f?make_float3(0,1,0):make_float3(1,0,0))));
			float3 v=cross(u,n);
			float3 d=normalize((u*cos(r1)+v*sin(r1))*sqrt(r2)+n*sqrt(1-r2));
			r=Ray(p,d);
		}
	}
	c[ti]=make_float4(cl);
	__syncthreads();
	
	for(int i=SAMPLE_PER_PIXCEL/2;i>0;i/=2){
		if(ti<i) c[ti]+=c[ti+i];
		__syncthreads();
	}
	if(ti==0){
//		d_c[idx]+=c[0]/SAMPLE_PER_PIXCEL;
		d_c[idx]+=c[0]/(SAMPLE_PER_PIXCEL*20);
	}
}

int main(){
	CHECK(hipSetDevice(0));
	
	//��ʼ��gpu��֡���� 
	CHECK(hipMalloc(&d_c,sizeof(h_c)));
	CHECK(hipMemset(d_c,0,sizeof(h_c)));
	//��ʼ��gpu���������� 
	CHECK(hipMalloc(&d_s,sizeof(h_s)));
	CHECK(hipMemcpy(d_s,h_s,sizeof(h_s),hipMemcpyHostToDevice));
	//��ʼ��gpu���������������״̬ 
	CHECK(hipMalloc(&states,sizeof(hiprandStateXORWOW_t)*SAMPLE_PER_PIXCEL));
	init_curand<<<1,SAMPLE_PER_PIXCEL>>>(states,time(0));
	
	//��cpu�����ɲ����㣨���ȷֲ��� 
	hiprandGenerator_t gen;
	checkCudaErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW));
	CHECK(hipMalloc(&d_samples,sizeof(float)*SAMPLE_PER_PIXCEL*2));
	checkCudaErrors(hiprandGenerateUniform(gen,d_samples,SAMPLE_PER_PIXCEL*2));
	
	Camera cam;
	cam.position=make_float3(50,52,215.6);
	cam.front=make_float3(0,0,-1);
	cam.right=make_float3(1,0,0);
	cam.up=make_float3(0,1,0);
	cam.fov=PI/4;
	
//for(int i=0;i<20;i++){

	castRay<<<dim3(WIDTH,HEIGHT),SAMPLE_PER_PIXCEL>>>(d_s,n,d_c,cam,d_samples,states);
	CHECK_KERNEL();
//}
	
	CHECK(hipMemcpy(h_c,d_c,sizeof(h_c),hipMemcpyDeviceToHost));
	
	CHECK(hipFree(states));
	CHECK(hipFree(d_c));
	CHECK(hipFree(d_s));
	
	//д��ͼƬ��ppm��ʽ��ֱ�ӽ�rgbֵд�룩 
	freopen("image.ppm","w",stdout);
	cout<<"P3\n"<<WIDTH<<' '<<HEIGHT<<' '<<255<<endl;
	for(int i=HEIGHT-1;i>=0;i--){
		for(int j=0;j<WIDTH;j++){
			float4 c=h_c[i][j];
			c=clamp(c,0.0f,1.0f);
			c=make_float4(pow(c.x,1/2.2),pow(c.y,1/2.2),pow(c.z,1/2.2),1);
			int4 tc=make_int4(c*make_float4(255));
			cout<<tc.x<<' '<<tc.y<<' '<<tc.z<<' ';
		}
	}
	return 0;
}
/*
nvcc -o test.exe -L "C:/Program Files/NVIDIA GPU Computing Toolkit/CUDA/v11.6/lib/x64" -l hiprand test.cu
*/
