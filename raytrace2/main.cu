#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cudaChecker.h"
#include <iostream>
#include "OBJ_Loader.hpp"
#include "Triangle.h"
#include "HitRecord.h"
#include "Matrial.h"
#include "Sobol.h"
#include "BRDF.h"
#include "Camera.h"
#include <vector>
#include <string>

#define WIDTH 1024
#define HEIGHT 768
#define SAMPLE_PER_PIXCEL 256

using namespace std;


vector<Triangle> h_tris;
Triangle* d_tris;

void AddObj(string f,Matrial mt);
void initScene();


float4 h_c[HEIGHT][WIDTH];
float4* d_c;

__device__
uint rhash(uint& seed){
	seed=seed^61^(seed>>16);
	seed*=9;
	seed=seed^(seed>>4);
	seed*=0x27d4eb2d;
    seed=seed^(seed>>15);
    return seed;
}
__device__
float2 rOffset(uint x,uint y,uint ti){
	uint seed=((x*1973+y*9277)+114514)|1;
	float u=rhash(seed)/4294967296.0f;
	float v=rhash(seed)/4294967296.0f;
    return make_float2(u,v);
}
__device__
float rSobol(int d,int i,float offset){
	float x=sobol(d,i)+offset;
	if(x>=1) x--;
	return x;
}
__device__
float3 SampleHemisphere(float x1,float x2){//������Ȳ��� 
	float z=x1;
	float r=sqrt(1-z*z);
	float phi=2*PI*x2;
	return make_float3(r*cos(phi),r*sin(phi),z);
}
__device__
float3 toNormalHemisphere(float3 v,float3 n){//��vͶӰ����nΪ����İ��� 
	float3 tp=make_float3(1,0,0);
	if(abs(n.x)>0.5) tp=make_float3(0,1,0);
	float3 t=normalize(cross(n,tp));
	float3 b=normalize(cross(n,t));
	return v.x*t+v.y*b+v.z*n;
}

__device__
bool closestHit(Ray r,Triangle* d_tris,int n,HitRecord& rec){
	bool hitted=false;
	float tmin=1e-4,tmax=1e10;
	for(int i=0;i<n;i++){
		bool t=hit(r,d_tris[i],tmin,tmax,rec);
		if(t) tmax=rec.t,hitted=true;
	}
	return hitted;
}

__global__
void kernel(Triangle* d_tris,int n,float4* d_c,Camera cam,uint T){
	int x=blockIdx.x,y=blockIdx.y;
	int idx=x+y*WIDTH;
	
	__shared__ float4 c[SAMPLE_PER_PIXCEL];
	int ti=threadIdx.x;
	float2 offset=rOffset(x,y,ti);
//	float2 offset=make_float2(0);
	
	int dep=0;
	float dx=rSobol(1,ti,offset.x),dy=rSobol(2,ti,offset.y);
	Ray r=cam.genRay(x+dx,y+dy,WIDTH,HEIGHT);
	float3 cl=make_float3(0.0f),cf=make_float3(1.0f);
	
	//printf("x:%d,y:%d,t:%d, dx:%f,dy:%f\n",x,y,ti,dx,dy);
	HitRecord rec;
	while(1){
		if(!closestHit(r,d_tris,n,rec)) break;
		
		Matrial& mt=rec.matrial;
		cl+=mt.emission*cf;
		if(++dep>6) break;
		
		float3 &p=rec.p;
		float3 &n=rec.normal;
		float3 v=-r.d;
		p+=n*0.1;
		
		float x1=rSobol(dep*2+0,T*1024+ti,offset.x);
		float x2=rSobol(dep*2+1,T*1024+ti,offset.y);
		//printf("x:%d,y:%d,t:%d, x1:%f,x2:%f\n",x,y,ti,x1,x2);
		float3 l=SampleHemisphere(x1,x2);
		l=toNormalHemisphere(l,n);
		
		float3 fr=BRDF(v,l,n,mt.baseColor,mt.metallic,mt.roughness);
		//float3 fr=mt.baseColor;
		//printf("%f, %f, %f\n",fr.x,fr.y,fr.z);
		cf*=fr*clamp(dot(n,l),0.0f,1.0f)*2*PI;
		r=Ray(p,l);
		
//		float3 H=normalize(n+l);
//		float3 F0=lerp(make_float3(0.04),mt.baseColor,mt.metallic);
//		cl=F_Schlick(F0,dot(v,n));
//		break;
	}
	c[ti]=make_float4(cl);
	__syncthreads();
	
	for(int i=SAMPLE_PER_PIXCEL/2;i>0;i/=2){
		if(ti<i) c[ti]+=c[ti+i];
		__syncthreads();
	}
	if(ti==0){
		d_c[idx]+=c[0]/(SAMPLE_PER_PIXCEL*40);
	}
}

int main(){
	CHECK(hipMalloc(&d_c,sizeof(h_c)));
	CHECK(hipMemset(d_c,0,sizeof(h_c)));
	
	initScene();
//	cout<<h_tris.size()<<endl;
	int tsz=sizeof(Triangle)*h_tris.size();
/*	HitRecord rec;
	int id=0;
	Ray r(make_float3(278,273,-800),make_float3(0,0,1));
	for(auto t:h_tris){
		printf("%d:",++id);
		if(hit(r,t,1e-4,1e20,rec)) printf("yes\n");
		printf("\n");
		printf("v0:(%f, %f, %f) ",t.v0.p.x,t.v0.p.y,t.v0.p.z);
		printf("v1:(%f, %f, %f) ",t.v1.p.x,t.v1.p.y,t.v1.p.z);
		printf("v2:(%f, %f, %f)\n",t.v2.p.x,t.v2.p.y,t.v2.p.z);
	}*/
	
	CHECK(hipMalloc(&d_tris,tsz));
	CHECK(hipMemcpy(d_tris,h_tris.data(),tsz,hipMemcpyHostToDevice));
	
	Camera cam;
	cam.position=make_float3(278,273,-800);
	cam.front=make_float3(0,0,1);
	cam.right=make_float3(-1,0,0);
	cam.up=make_float3(0,1,0);
	cam.fov=PI/4;

for(int i=1;i<=40;i++){

	kernel<<<dim3(WIDTH,HEIGHT),SAMPLE_PER_PIXCEL>>>(d_tris,h_tris.size(),d_c,cam,i);
	CHECK_KERNEL();
}
	
	
	CHECK(hipMemcpy(h_c,d_c,sizeof(h_c),hipMemcpyDeviceToHost));
	
	//д��ͼƬ��ppm��ʽ��ֱ�ӽ�rgbֵд�룩 
	freopen("image2.ppm","w",stdout);
	cout<<"P3\n"<<WIDTH<<' '<<HEIGHT<<' '<<255<<endl;
	for(int i=HEIGHT-1;i>=0;i--){
		for(int j=0;j<WIDTH;j++){
			float4 c=h_c[i][j];
			c=clamp(c,0.0f,1.0f);
			//c=c/(c+1.0);
			c=make_float4(pow(c.x,1/2.2),pow(c.y,1/2.2),pow(c.z,1/2.2),1);
			int4 tc=make_int4(c*255);
			cout<<tc.x<<' '<<tc.y<<' '<<tc.z<<' ';
		}
	}
	
	return 0;
}


void AddObj(string f,Matrial mt){
	objl::Loader loader;
    loader.LoadFile(f);
    auto mesh=loader.LoadedMeshes[0];
    for(int i=0;i<mesh.Vertices.size();i+=3){
    	auto v0=mesh.Vertices[i+0].Position;
    	auto v1=mesh.Vertices[i+1].Position;
    	auto v2=mesh.Vertices[i+2].Position;
    	
    	Triangle tri;
    	tri.v0.p=make_float3(v0.X,v0.Y,v0.Z);
    	tri.v1.p=make_float3(v1.X,v1.Y,v1.Z);
    	tri.v2.p=make_float3(v2.X,v2.Y,v2.Z);
    	tri.matrial=mt;
    	
    	h_tris.push_back(tri);
	}
}

void initScene(){
	Matrial nr_write={make_float3(0.8),0,1,make_float3(0)};
	Matrial mr_write={make_float3(0.8),1,0.5,make_float3(0)};
	
	Matrial ns_write={make_float3(0.8),0,0.1,make_float3(0)};
	Matrial ms_write={make_float3(0.8),1,0.1,make_float3(0)};
	
	Matrial mr_blue={make_float3(0.35,0.35,1.0),1,1,make_float3(0)};
	Matrial ms_blue={make_float3(0.35,0.35,1.0),1,0.1,make_float3(0)};
	
	Matrial nr_blue={make_float3(0.35,0.35,1.0),0,1,make_float3(0)};
	Matrial ns_blue={make_float3(0.35,0.35,1.0),0,0.1,make_float3(0)};
	
	Matrial mr_red={make_float3(1.0,0.35,0.35),1,1,make_float3(0)};
	Matrial ms_red={make_float3(1.0,0.35,0.35),1,0.1,make_float3(0)};
	
	Matrial nr_red={make_float3(1.0,0.35,0.35),0,1,make_float3(0)};
	Matrial ns_red={make_float3(1.0,0.35,0.35),0,0.1,make_float3(0)};
	
	Matrial mr_yellow={make_float3(1.0,1.0,0.5),1,1,make_float3(0)};
	
	Matrial mr_ori={make_float3(1.0,0.5,0.3),1,1,make_float3(0)};
	
	Matrial light={make_float3(0),1,1,make_float3(30)};
	
	AddObj("cornellbox/floor.obj",mr_write);
	AddObj("cornellbox/shortbox.obj",mr_write);
	AddObj("cornellbox/tallbox.obj",mr_write);
	AddObj("cornellbox/left.obj",nr_red);
	AddObj("cornellbox/right.obj",nr_blue);
	AddObj("cornellbox/light.obj",light);
}
