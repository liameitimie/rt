#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include "cudaChecker.h"
using namespace std;
//using namespace cv;


int main(){
	//cv::Mat image = cv::imread("C:/BaiduNetdiskDownload/����pbr��ͼ/����ʯ/Marble021/Marble021_4K_NormalDX.png", cv::IMREAD_ANYCOLOR | cv::IMREAD_ANYDEPTH);
	cv::Mat image = cv::imread("C:/BaiduNetdiskDownload/����pbr��ͼ/����ʯ/Marble021/Marble021_4K_Roughness.png", cv::IMREAD_ANYCOLOR | cv::IMREAD_ANYDEPTH);
	//cv::Mat image=cv::imread("image.png"/*,IMREAD_ANYCOLOR|IMREAD_ANYDEPTH*/);
	//cv::cvtColor(image, image, cv::COLOR_BGR2RGB);
	
	int w=image.cols;
	int h=image.rows;
	int imtype=image.type();
	int c=(imtype>>3)+1;//��ɫͨ���� (channel) 
	int t=(imtype&0x07);//opencv��ɫ��ʽ 0:CV_8U, 1:CV8S, 2:CV_16U, 3:CV_16S, 4:CV_32S, 5:CV_32F, 6:CV_64F
	int b=(8<<(t>>1));  //ÿ����ɫ��λ�� 
	
//	cout<<w<<' '<<h<<' '<<imtype<<' '<<c<<' '<<t<<' '<<b<<endl;
//	cout<<image.isContinuous()<<endl;
	
	if(c==3){
		cv::cvtColor(image, image, cv::COLOR_RGB2RGBA);
		imtype=image.type();
		c=(imtype>>3)+1;
		t=(imtype&0x07);
		b=(8<<(t>>1));
	}
	uchar* pdata=image.data;
	
	cout<<w<<' '<<h<<' '<<imtype<<' '<<c<<' '<<t<<' '<<b<<endl;
	cout<<image.isContinuous()<<endl;
	
//	cout<<(int)pdata[0]<<' '<<(int)pdata[1]<<' '<<(int)pdata[2]<<endl;
	
	/*������ɫͨ����ʽ*/
//	hipChannelFormatDesc cfd=hipCreateChannelDesc<float2>();
	//hipChannelFormatDesc cfd=hipCreateChannelDesc(8,8,8,0,hipChannelFormatKindUnsigned);
//	hipChannelFormatDesc cfd=hipCreateChannelDesc(
//		(c>=1)?b:0, (c>=2)?b:0, (c>=3)?b:0, (c>=4)?b:0,
//		((t>4)? hipChannelFormatKindFloat
//			:(((t&1)||t==4)? hipChannelFormatKindSigned
//				:hipChannelFormatKindUnsigned
//			)
//		)
//	);
//	cfd.x=(c>=1)?b:0,cfd.y=(c>=2)?b:0,cfd.z=(c>=3)?b:0,cfd.w=(c>=4)?b:0;
//	if(t>4) cfd.f=hipChannelFormatKindFloat;
//	else{
//		if(t&1) cfd.f=hipChannelFormatKindSigned;
//		else cfd.f=hipChannelFormatKindUnsigned;
//	}
	
//	cout<<cfd.x<<' '<<cfd.y<<' '<<cfd.z<<' '<<cfd.w<<' '<<cfd.f<<endl;
//	
//	hipArray_t a;
//	CHECK(hipMallocArray(&a,&cfd,w,h));
	
//	cv::Vec3w cc=image.at<cv::Vec3w>(0,0);
//	cout<<(int)cc[0]<<' '<<(int)cc[1]<<' '<<(int)cc[2]<<endl;
	//imwrite("image.png",image);
//	int t=image.type();
//	cout<<t<<' '<<(t>>3)<<' '<<(t&0x07)<<endl;
	return 0;
}
/*
nvcc -L C:/opencv/build/x64/vc15/lib -l opencv_world454 -I C:/opencv/build/include -I C:/opencv/build/include/opencv2 -o test.exe test.cu
*/
