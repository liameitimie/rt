#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include "cudaChecker.h"
#include "myMath.h"
using namespace std;
//using namespace cv;

#define WIDTH 4096
#define HEIGHT 2048

float4 h_c[HEIGHT][WIDTH];
float4* d_c;

__global__
void mycopy(float4* d_c,hipTextureObject_t tex){
	int x=blockIdx.x,y=blockIdx.y;
	int idx=x+y*WIDTH;
	float u=(float)x/WIDTH,v=(float)y/HEIGHT;
	
	d_c[idx]=tex2D<float4>(tex,u,v);
}

float3 ACESFilm(float3 x){
    float a = 2.51f;
    float b = 0.03f;
    float c = 2.43f;
    float d = 0.59f;
    float e = 0.14f;
    return clamp((x*(a*x+b))/(x*(c*x+d)+e),0.0f,1.0f);
}

int main(){
	//cv::Mat image = cv::imread("C:/BaiduNetdiskDownload/����pbr��ͼ/����ʯ/Marble021/Marble021_4K_NormalDX.png", cv::IMREAD_ANYCOLOR | cv::IMREAD_ANYDEPTH);
	//cv::Mat image = cv::imread("C:/BaiduNetdiskDownload/����pbr��ͼ/����ʯ/Marble021/Marble021_4K_Roughness.png", cv::IMREAD_ANYCOLOR | cv::IMREAD_ANYDEPTH);
	//cv::Mat image=cv::imread("image2.png"/*,IMREAD_ANYCOLOR|IMREAD_ANYDEPTH*/);
	
	//cv::Mat image=cv::imread("C:/BaiduNetdiskDownload/����pbr��ͼ/hdr������/lebombo_4k.hdr",cv::IMREAD_ANYCOLOR|cv::IMREAD_ANYDEPTH);
	//cv::Mat image=cv::imread("C:/BaiduNetdiskDownload/����pbr��ͼ/hdr������/the_sky_is_on_fire_4k.hdr",cv::IMREAD_ANYCOLOR|cv::IMREAD_ANYDEPTH);
	cv::Mat image=cv::imread("C:/BaiduNetdiskDownload/����pbr��ͼ/hdr������/studio_garden_4k.hdr",cv::IMREAD_ANYCOLOR|cv::IMREAD_ANYDEPTH);
	
	//cv::imwrite("image2.png",image);
	
	int w=image.cols;
	int h=image.rows;
	//if(!w&&!h) return 0; 
	int imtype=image.type();
	if((imtype>>3)+1==3){//3ͨ����cuda����֧��3ͨ��
		cv::cvtColor(image,image,cv::COLOR_BGR2RGBA);
		imtype=image.type();
	}
	int c=(imtype>>3)+1;//��ɫͨ���� (channel) 
	int t=(imtype&0x07);//opencv��ɫ��ʽ 0:CV_8U, 1:CV8S, 2:CV_16U, 3:CV_16S, 4:CV_32S, 5:CV_32F, 6:CV_64F
	int b=(8<<(t>>1));  //ÿ����ɫͨ����λ�� 
	
	int sz=w*h*c*(b>>3);
	uchar* pdata=image.data;
	
	cout<<image.size()<<' '<<sz<<endl;
	cout<<w<<' '<<h<<' '<<imtype<<' '<<c<<' '<<t<<' '<<b<<endl;
	cout<<image.isContinuous()<<endl;
	
	auto color=image.at<cv::Vec4f>(0,0);
	cout<<color[0]<<' '<<color[1]<<' '<<color[2]<<' '<<color[3]<<endl;
	
	for(int i=0;i<16;i+=4){
		float t;
		uchar* p=(uchar*)&t;
		for(int j=0;j<4;j++) p[j]=pdata[i+j];
		cout<<t<<endl;
		//cout<<(int)(pdata[0+i*1]/*+pdata[1+i*2]*256*/)<<' ';
	}cout<<endl;
	
	
	hipChannelFormatDesc cfd=hipCreateChannelDesc(
		(c>=1)?b:0, (c>=2)?b:0, (c>=3)?b:0, (c>=4)?b:0,
		((t>4)? hipChannelFormatKindFloat
			:(((t&1)||t==4)? hipChannelFormatKindSigned
				:hipChannelFormatKindUnsigned
			)
		)
	);
	
	cout<<cfd.x<<' '<<cfd.y<<' '<<cfd.z<<' '<<cfd.w<<' '<<cfd.f<<endl;
	
	//����cuda���鲢����ͼ��cuda���� 
	hipArray_t a;
	CHECK(hipMallocArray(&a,&cfd,w,h));
	CHECK(hipMemcpyToArray(a,0,0,pdata,sz,hipMemcpyHostToDevice));
	
	//������Դ���� 
	hipResourceDesc rd;                //��Դ������������Դ����������Դ��ַ 
    memset(&rd,0,sizeof(rd));
    rd.resType=hipResourceTypeArray;   //��Դ����Ϊcuda���飬 
    rd.res.array.array=a;               //����֮�⻹��mipmap��linear��pitch2D 
    
    //������������
	hipTextureDesc td;                 //����������������ֵ�������������� 
	memset(&td,0,sizeof(td));
	td.normalizedCoords=true;           //���ʵ�uv�����Թ�һ��ģʽ���� 
	td.filterMode=hipFilterModeLinear; //��ֵ����Ϊ���Բ�ֵ��0:�����, 1:���� 
	//td.readMode=hipReadModeNormalizedFloat;//��ȡ�����ݹ�һ�� 
	td.readMode=hipReadModeElementType;
	//td.addressMode[0..2] //��uv����һʱ��ȡ�Ĳ��ԣ�0:wrap, 1:clamp, 2:mirror, 3:border  
	
	//����������� 
	hipTextureObject_t tex=0;
	CHECK(hipCreateTextureObject(&tex,&rd,&td,NULL));
	
	CHECK(hipMalloc(&d_c,sizeof(h_c)));
	
	mycopy<<<dim3(WIDTH,HEIGHT),1>>>(d_c,tex);
	
	CHECK(hipMemcpy(h_c,d_c,sizeof(h_c),hipMemcpyDeviceToHost));
	
	//д��ͼƬ��ppm��ʽ��ֱ�ӽ�rgbֵд�룩 
	freopen("image5.ppm","w",stdout);
	cout<<"P3\n"<<WIDTH<<' '<<HEIGHT<<' '<<255<<endl;
	for(int i=0;i<HEIGHT;i++){
		for(int j=0;j<WIDTH;j++){
			float3 c=make_float3(h_c[i][j]);
			c=ACESFilm(c);
			int r=c.x*255;
			int g=c.y*255;
			int b=c.z*255;
			cout<<r<<' '<<g<<' '<<b<<' ';
		}
	}
	
	return 0;
}
/*
nvcc -L C:/opencv/build/x64/vc15/lib -l opencv_world454 -I C:/opencv/build/include -I C:/opencv/build/include/opencv2 -o test2.exe test2.cu
*/
