#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
//#include "hiprand.h"
//#include "hiprand/hiprand_kernel.h"
#include "cudaChecker.h"

#include <iostream>
#include "myMath.h"
#include "Ray.h"
#include "Sphere.h"
#include "Camera.h"
#include "Matrial.h"
#include "Sampler.h"

using namespace std;

#define WIDTH 1024
#define HEIGHT 768
#define SAMPLE_PER_PIXCEL 1024
//#define PI 3.141592653589793f

Matrial light={make_float3(0), 0, 0, make_float3(12,12,12)};

Matrial nr_black={make_float3(0.1,0.1,0.1), 0, 1, make_float3(0)};
Matrial nr_write={make_float3(1.0,1.0,1.0), 0, 1, make_float3(0)};
Matrial nr_blue={make_float3(0.35,0.35,1.0), 0, 1, make_float3(0)};
Matrial nr_red={make_float3(1.0,0.35,0.35), 0, 1, make_float3(0)};

Matrial ms_blue={make_float3(0.35,0.35,1.0), 1, 0.2, make_float3(0)};
Matrial ns_red={make_float3(1.0,0.35,0.35), 0, 0, make_float3(0)};

Matrial mr_write={make_float3(1.0,1.0,1.0), 1, 0.2, make_float3(0)};
Matrial ms_write={make_float3(1.0,1.0,1.0), 1, 0, make_float3(0)};

Matrial mr1={make_float3(1.0,1.0,1.0), 0, 0, make_float3(0)};
Matrial mr2={make_float3(1.0,1.0,1.0), 0, 0.25, make_float3(0)};
Matrial mr3={make_float3(1.0,1.0,1.0), 0, 0.5, make_float3(0)};
Matrial mr4={make_float3(1.0,1.0,1.0), 0, 0.75, make_float3(0)};
Matrial mr5={make_float3(1.0,1.0,1.0), 0, 1, make_float3(0)};

Sphere* d_s;
//Sphere h_s[]={
//	Sphere(16.5,make_float3(27,16.5,47),mr1),
//	Sphere(16.5,make_float3(73,16.5,78),mr2),
//	Sphere(16.5,make_float3(27,55,47),mr3),
//	Sphere(16.5,make_float3(73,55,78),mr4),
//	Sphere(16.5,make_float3(27,93,47),mr5)
//};
Sphere h_s[]={
	//    �뾶  λ��                           ���� 
	Sphere(1e5, make_float3( 1e5+1,40.8,81.6), ms_blue),//��ǽ�� 
	Sphere(1e5, make_float3(-1e5+99,40.8,81.6),ns_red),//��ǽ�� 
	Sphere(1e5, make_float3(50,40.8, 1e5),     nr_write),//��ǽ�� 
	Sphere(1e5, make_float3(50,40.8,-1e5+250), nr_black),//ǰǽ�� 
	Sphere(1e5, make_float3(50, 1e5, 81.6),    nr_write),//�ذ�
	Sphere(1e5, make_float3(50,-1e5+81.6,81.6),nr_write),//�컨��
	Sphere(16.5,make_float3(27,16.5,47),       mr_write),//������ 
	Sphere(16.5,make_float3(73,16.5,78),       nr_write),//�������� 
	Sphere(600, make_float3(50,681.6-.27,81.6),light) //��Դ
};

int n=sizeof(h_s)/sizeof(Sphere);

float4 h_c[HEIGHT][WIDTH];
float4* d_c;

__device__
bool closestHit(const Ray& r,Sphere* d_s,int n,float& t,int& id){
	bool hitted=false;
	float tmin=1e-4,tmax=1e10;
	for(int i=0;i<n;i++){
		float tmp=intersect(d_s[i],r,tmin,tmax);
		if(tmp>0){
			hitted=true,tmax=tmp;
			t=tmp,id=i;
		}
	}
	return hitted;
}

__global__
void castRay(Sphere* d_s,int n,float4* d_c,Camera cam,int T){
	int x=blockIdx.x,y=blockIdx.y;
	int idx=x+y*WIDTH;
	
	float xi[3],offset[3];//���������ƫ���� 
	rOffset(offset,3,x,y); 
	
	__shared__ float4 c[SAMPLE_PER_PIXCEL];
	int ti=threadIdx.x;
	
	rSobol(xi,2,1,T*SAMPLE_PER_PIXCEL+ti,offset);
	Ray r=cam.genRay(x+xi[0],y+xi[1],WIDTH,HEIGHT);
	float3 cl=make_float3(0.0f),cf=make_float3(1);
	int dep=0;
	while(1){
		float t;int id;
		if(!closestHit(r,d_s,n,t,id)){/*cl=make_float3(0.5);*/break;}
		Sphere obj=d_s[id];
		
		Matrial& mt=obj.matrial;
		cl+=cf*mt.emission;
		//cf*=obj.c;
		if(++dep>6) break;
		
		float3 p=r.o+r.d*t;
		float3 tn=normalize(p-obj.p);
		float3 n=dot(tn,r.d)<0?tn:-tn;
		p+=n*make_float3(1e-1);
		float3 v=-r.d;
		
		rSobol(xi,3,dep*3,T*SAMPLE_PER_PIXCEL+ti,offset);
		
//		float3 l=SpUniformH(xi_1,xi_2,n);
//		float pdf=PdfUniformH(l,n);
//		float3 l=SpCosH(xi[0],xi[1],n);
//		float pdf=PdfCosH(l,n);
		float3 l=SpBRDF(xi[0],xi[1],xi[2],v,n,mt.metallic,mt.roughness);
		float pdf=PdfBRDF(l,v,n,mt.metallic,mt.roughness);
		
		if(pdf<1e-4) break;
		
		float3 fr=BRDF(v,l,n,mt.baseColor,mt.metallic,mt.roughness);
//		float3 fr=BRDF(l,v,n,mt.baseColor,mt.metallic,mt.roughness);
		
		cf*=fr*dot(n,l)/pdf;
		
		r=Ray(p,l);
		
		
//		cl=cf*0.5;
//		break;
	}
//	if(cl.x<0||cl.y<0||cl.z<0) cl=make_float3(1);
	c[ti]=make_float4(cl);
	__syncthreads();
	
	for(int i=SAMPLE_PER_PIXCEL/2;i>0;i/=2){
		if(ti<i) c[ti]+=c[ti+i];
		__syncthreads();
	}
	if(ti==0){
//		d_c[idx]+=c[0]/SAMPLE_PER_PIXCEL;
		d_c[idx]+=c[0]/(SAMPLE_PER_PIXCEL*20);
	}
}

int main(){
	CHECK(hipSetDevice(0));
	
	//��ʼ��gpu��֡���� 
	CHECK(hipMalloc(&d_c,sizeof(h_c)));
	CHECK(hipMemset(d_c,0,sizeof(h_c)));
	//��ʼ��gpu���������� 
	CHECK(hipMalloc(&d_s,sizeof(h_s)));
	CHECK(hipMemcpy(d_s,h_s,sizeof(h_s),hipMemcpyHostToDevice));
	
	Camera cam;
	cam.position=make_float3(50,52,215.6);
	cam.front=make_float3(0,0,-1);
	cam.right=make_float3(1,0,0);
	cam.up=make_float3(0,1,0);
	cam.fov=PI/4;
	
for(int i=0;i<20;i++){

	castRay<<<dim3(WIDTH,HEIGHT),SAMPLE_PER_PIXCEL>>>(d_s,n,d_c,cam,i);
	CHECK_KERNEL();
}
	
	CHECK(hipMemcpy(h_c,d_c,sizeof(h_c),hipMemcpyDeviceToHost));
	
	//CHECK(hipFree(states));
	CHECK(hipFree(d_c));
	CHECK(hipFree(d_s));
	
	//д��ͼƬ��ppm��ʽ��ֱ�ӽ�rgbֵд�룩 
	freopen("image1.ppm","w",stdout);
	cout<<"P3\n"<<WIDTH<<' '<<HEIGHT<<' '<<255<<endl;
	for(int i=HEIGHT-1;i>=0;i--){
		for(int j=0;j<WIDTH;j++){
			float4 c=h_c[i][j];
			c=clamp(c,0.0f,1.0f);
			c=make_float4(pow(c.x,1/2.2),pow(c.y,1/2.2),pow(c.z,1/2.2),1);
			int4 tc=make_int4(c*make_float4(255));
			cout<<tc.x<<' '<<tc.y<<' '<<tc.z<<' ';
		}
	}
	return 0;
}
/*
nvcc -o test.exe -L "C:/Program Files/NVIDIA GPU Computing Toolkit/CUDA/v11.6/lib/x64" -l hiprand test.cu
*/
