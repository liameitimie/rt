#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "cudaChecker.h"
#include "myMath.h"
#include "Sobol.h"
#include "Sphere.h"
#include "Camera.h"
#include "Sampler.h"

using namespace std;

#define WIDTH 1024
#define HEIGHT 768
#define SAMPLE_PER_PIXCEL 1024

float4 h_c[HEIGHT][WIDTH];
float4* d_c;

Sphere* d_s;
Sphere h_s[]={
	//    �뾶  λ��                           �Է���          ��ɫ                     ���� 
	Sphere{1e5, make_float3( 1e5+1,40.8,81.6), make_float3(0), make_float3(1.0,0.35,0.35),1},//��ǽ�� 
	Sphere{1e5, make_float3(-1e5+99,40.8,81.6),make_float3(0), make_float3(0.35,0.35,1.0),1},//��ǽ�� 
	Sphere{1e5, make_float3(50,40.8, 1e5),     make_float3(0), make_float3(1.0,1.0,1.0),1},//��ǽ�� 
	Sphere{1e5, make_float3(50,40.8,-1e5+250), make_float3(0), make_float3(0),          1},//ǰǽ�� 
	Sphere{1e5, make_float3(50, 1e5, 81.6),    make_float3(0), make_float3(1.0,1.0,1.0),1},//�ذ�
	Sphere{1e5, make_float3(50,-1e5+81.6,81.6),make_float3(0), make_float3(1.0,1.0,1.0),1},//�컨��
	Sphere{16.5,make_float3(27,16.5,47),       make_float3(0), make_float3(1.0,1.0,1.0),0},//�� 
	Sphere{16.5,make_float3(73,16.5,78),       make_float3(0), make_float3(1.0,1.0,1.0),0},//�� 
	Sphere{600, make_float3(50,681.6-.27,81.6),make_float3(12,12,12),   make_float3(0), 1} //��Դ
};
int n=sizeof(h_s)/sizeof(Sphere);


__device__
bool closestHit(const Ray& r,Sphere* d_s,int n,float& t,int& id){
	bool hitted=false;
	float tmin=1e-4,tmax=1e10;
	for(int i=0;i<n;i++){
		float tmp=intersect(d_s[i],r,tmin,tmax);
		if(tmp>0){
			hitted=true,tmax=tmp;
			t=tmp,id=i;
		}
	}
	return hitted;
}

__device__
float3 RTMain(Ray r,Sphere* d_s,int n,int maxdep,int ti,float* xi,float* offset){
	float3 sL={0,0,0},sK={1,1,1};
	int dep=0;
	while(1){
		float t;int id;
		if(!closestHit(r,d_s,n,t,id)) break;
		Sphere& obj=d_s[id];
		
		sL+=sK*obj.e;
		if(++dep>maxdep) break;
		
		float a=obj.roughness*obj.roughness;
		rSobol(xi,2,dep*2,ti,offset);
		
		float3 p=r.o+r.d*t;
		float3 n=normalize(p-obj.p);
		float3 v=-r.d;
		float3 l;
		float IOR=1.5;
		if(dot(n,v)<=0) n=-n,IOR=-IOR;
		
		if(a>0.9){
			p+=n*0.1;
			l=SpCosH(xi[0],xi[1],n);
			//sK*=c/Pi*dot(n,l)/PdfCosH(l,n);
			sK*=obj.c;
		}
		else if(a<0.001){
			float F;
			
			float VoH=dot(v,n);
			float cosi=VoH;
			float sini=sqrt(1-cosi*cosi);
			float sint=1/IOR*sini;
			float cost;
			if(sint>=1) F=1;
			else{
				cost=sqrt(1-sint*sint);
				float rl=(IOR*cosi-cost)/(IOR*cosi+cost);
				float rp=(cosi-IOR*cost)/(cosi+IOR*cost);
				F=0.5*(rl*rl+rp*rp);
			}
			
			if(xi[0]<F){
				p+=n*0.1;
				l=(2*dot(n,v))*n-v;
			}
			else{
				p+=n*-0.1;
				l=-(1/IOR)*v+((1/IOR)*cosi-cost)*n;
			}
		}
		else{
			float3 h=SpGGX(xi[0],xi[1],n,a);
			
		}
		r.o=p,r.d=l;
	}
	return sL;
}

__global__
void RTKernel(Sphere* d_s,int n,float4* d_c,Camera cam){
	int x=blockIdx.x,y=blockIdx.y;
	int idx=x+y*WIDTH;
	__shared__ float4 c[SAMPLE_PER_PIXCEL];
	int ti=threadIdx.x;
	
	float xi[3],offset[3];
	rOffset(offset,3,x,y);
	rSobol(xi,2,1,233,offset);
	Ray r=cam.genRay(x+xi[0],y+xi[1],WIDTH,HEIGHT);
	
	float3 radiance=RTMain(r,d_s,n,12,ti,xi,offset);
	c[ti]=make_float4(radiance);
	__syncthreads();
	
	for(int i=SAMPLE_PER_PIXCEL/2;i>0;i/=2){
		if(ti<i) c[ti]+=c[ti+i];
		__syncthreads();
	}
	if(ti==0) d_c[idx]+=c[0]/SAMPLE_PER_PIXCEL;
}

int main(){
	CHECK(hipMalloc(&d_c,sizeof(h_c)));
	CHECK(hipMemset(d_c,0,sizeof(h_c)));
	CHECK(hipMalloc(&d_s,sizeof(h_s)));
	CHECK(hipMemcpy(d_s,h_s,sizeof(h_s),hipMemcpyHostToDevice));
	
	Camera cam;
	cam.position=make_float3(50,52,215.6);
	cam.front=make_float3(0,0,-1);
	cam.right=make_float3(1,0,0);
	cam.up=make_float3(0,1,0);
	cam.fov=PI/4;
	
	RTKernel<<<dim3(WIDTH,HEIGHT),SAMPLE_PER_PIXCEL>>>(d_s,n,d_c,cam);
	CHECK_KERNEL();
	
	CHECK(hipMemcpy(h_c,d_c,sizeof(h_c),hipMemcpyDeviceToHost));
	
	freopen("image.ppm","w",stdout);
	cout<<"P3\n"<<WIDTH<<' '<<HEIGHT<<' '<<255<<endl;
	for(int i=HEIGHT-1;i>=0;i--){
		for(int j=0;j<WIDTH;j++){
			float4 c=h_c[i][j];
			c=clamp(c,0.0f,1.0f);
			c=make_float4(pow(c.x,1/2.2),pow(c.y,1/2.2),pow(c.z,1/2.2),1);
			int4 tc=make_int4(c*make_float4(255));
			cout<<tc.x<<' '<<tc.y<<' '<<tc.z<<' ';
		}
	}
	
	return 0;
}
