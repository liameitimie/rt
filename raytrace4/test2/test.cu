#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "cudaChecker.h"

#include "Sampler.h"
#include "object.h"
#include "BSDF.h"
#include "TextureManager.h"
#include "Camera.h"

using namespace std;

#define WIDTH 1024
#define HEIGHT 768
#define SAMPLE_PER_PIXCEL 512

float4 h_c[HEIGHT][WIDTH];
float4* d_c;

object* d_s;
object h_s[2];
int n=2;

void initScene(){
	h_s[0].objectType=object::sphere;
	h_s[0].p={0,1,0};
	h_s[0].r=1;
	h_s[0].matrial.baseColor={1.022,0.782,0.344};
	h_s[0].matrial.metallic=1;
	h_s[0].matrial.roughness=0.2;
	h_s[0].matrial.IOR=1.5;
	h_s[0].matrial.specTrans=0;
	
	h_s[1].objectType=object::triangle;
	h_s[1].p1={200,0,-100};
	h_s[1].p2={-100,0,-100};
	h_s[1].p3={-100,0,200};
	h_s[1].matrial.baseColor={0.5,0.5,0.5};
	h_s[1].matrial.roughness=1;
	h_s[1].matrial.IOR=1.5;
	h_s[1].matrial.specTrans=0;
}

__device__
bool closestHit(Ray r,object* d_s,int n,float& t,int& id){
	bool hitted=false;
	float tmin=1e-4,tmax=1e10;
	for(int i=0;i<n;i++){
		float tmp=hit(r,d_s[i],tmin,tmax);
		if(tmp>0){
			hitted=true,tmax=tmp;
			t=tmp,id=i;
		}
	}
	return hitted;
}

__global__
void RTKernel(object* d_s,int n,float4* d_c,Camera cam,hipTextureObject_t tex){
	int x=blockIdx.x,y=blockIdx.y;
	int idx=x+y*WIDTH;
	__shared__ float4 c[SAMPLE_PER_PIXCEL];
	int ti=threadIdx.x;
	
	float xi[3],offset[3];
	rOffset(offset,3,x,y);
	
	float3 color={0,0,0};
	
for(int i=1;i<=20;i++){

	rSobol(xi,2,1,i*SAMPLE_PER_PIXCEL+ti,offset);
	Ray r=cam.genRay(x+xi[0],y+xi[1],WIDTH,HEIGHT);
	
	float3 sL={0,0,0},sK={1,1,1};
	int dep=0;
	while(1){
		float t;int id;
		if(!closestHit(r,d_s,n,t,id)){
			float v=acos(r.d.y)/PI;
			float u=0;
			if(v>0) u=atan2(r.d.z,r.d.x)/(2*PI)+0.2;
			float4 c=tex2D<float4>(tex,u,v);
			sL+=sK*make_float3(c);
			break;
		}
		object& obj=d_s[id];
		
		if(++dep>12) break;
		
		float3 p=r.o+r.d*t;
		float3 n=CalcNormal(obj,p);
		float3 v=-r.d;
		
//		sL=n*0.5+0.5;break;
		
		rSobol(xi,3,dep*3,i*SAMPLE_PER_PIXCEL+ti+1,offset);
		
		BSDFContext c;
		InitBSDFContext(c,p,n,v,obj.matrial);
		float3 K=SampleBSDF(c,xi[0],xi[1],xi[2]);
		
		sK*=K;
		if(length(sK)==0) break;
		
		r.o=c.P,r.d=c.L;
	}
	if(length(sK)<1e3) color+=sL/20;
}
	c[ti]=make_float4(color);
	__syncthreads();
	
	for(int i=SAMPLE_PER_PIXCEL/2;i>0;i/=2){
		if(ti<i) c[ti]+=c[ti+i];
		__syncthreads();
	}
	if(ti==0) d_c[idx]+=c[0]/SAMPLE_PER_PIXCEL;
}

float3 ACESFilm(float3 x){
    float a = 2.51f;
    float b = 0.03f;
    float c = 2.43f;
    float d = 0.59f;
    float e = 0.14f;
    return clamp((x*(a*x+b))/(x*(c*x+d)+e),0.0f,1.0f);
}

int main(){
/*	for(float a=0;a<1;a+=0.1){
		printf("a:%f\n",a);
		for(float NoL=0;NoL<1;NoL+=0.1){
			for(float NoV=0;NoV<1;NoV+=0.1){
				printf("%f ",V_SmithGGXCorrelated(a,NoL,NoV));
			}
			printf("\n");
		}
	}*/
	
	initScene();
	
	CHECK(hipMalloc(&d_c,sizeof(h_c)));
	CHECK(hipMemset(d_c,0,sizeof(h_c)));
	CHECK(hipMalloc(&d_s,sizeof(h_s)));
	CHECK(hipMemcpy(d_s,h_s,sizeof(h_s),hipMemcpyHostToDevice));
	
	Camera cam;
	cam.position=make_float3(0,1.5,5);
	cam.front=make_float3(0,0,-1);
	cam.right=make_float3(1,0,0);
	cam.up=make_float3(0,1,0);
	cam.fov=PI/4;
	
	hipTextureObject_t tex=textureManager.texture("C:/BaiduNetdiskDownload/����pbr��ͼ/hdr������/lebombo_4k.hdr");
	
	RTKernel<<<dim3(WIDTH,HEIGHT),SAMPLE_PER_PIXCEL>>>(d_s,n,d_c,cam,tex);
	CHECK_KERNEL();
	
	CHECK(hipMemcpy(h_c,d_c,sizeof(h_c),hipMemcpyDeviceToHost));
	
	freopen("image.ppm","w",stdout);
	cout<<"P3\n"<<WIDTH<<' '<<HEIGHT<<' '<<255<<endl;
	for(int i=HEIGHT-1;i>=0;i--){
		for(int j=0;j<WIDTH;j++){
			float3 c=make_float3(h_c[i][j]);
			//c=ACESFilm(c);
			c=clamp(c,0,1);
			c=make_float3(pow(c.x,1/2.2),pow(c.y,1/2.2),pow(c.z,1/2.2));
			int3 tc=make_int3(c*255);
			cout<<tc.x<<' '<<tc.y<<' '<<tc.z<<' ';
		}
	}
	
	return 0;
}
/*
nvcc -L C:/opencv/build/x64/vc15/lib -l opencv_world454 -I C:/opencv/build/include -I C:/opencv/build/include/opencv2 -o test.exe test.cu
*/
