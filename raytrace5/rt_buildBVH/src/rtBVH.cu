#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <vector>
#include <queue>
#include "rt_api.h"
#include "hip/hip_vector_types.h"
#include "myMath.h"

using std::vector;
using std::queue;

extern "C" 
void rtGetAccelerationStructurePrebuildInfo(
	rtAccelerationStructureBuildInputs* pInputs,
	rtAccelerationStructurePrebuildInfo* pInfo
){
	
}

/*
node:
	float4 x:c0minx y:c0maxx z:c0miny w:c0maxy
	float4 x:c1minx y:c1maxx z:c1miny w:c1maxy
	float4 x:c0minz y:c0maxz z:c1minz w:c1maxz
	float4 x(int):c0-nodeid|~c0-triid y(int):c1-nodeid|~c1-triid
	
woopifiedtri: (Woop's unit triangle intersection test [Woop 2004])
	float4 v0
	float4 v1
	float4 v2
	... other tri
	float4 x:0x80000000(-0f)  terminator

*/

struct aabb
{
	float3 pmin;
	float3 pmax;
	aabb(){pmin=make_float3(1e18),pmax=make_float3(-1e18);}
	aabb(float3 p){pmin=p,pmax=p;}
	aabb(float3 p1,float3 p2){
		pmin=make_float3(min(p1.x,p2.x),min(p1.y,p2.y),min(p1.z,p2.z));
		pmax=make_float3(max(p1.x,p2.x),max(p1.y,p2.y),max(p1.z,p2.z));
	}
	aabb(float3 p1,float3 p2,float3 p3){
		*this=merge(aabb(p1,p2),p3);
	}
	friend aabb merge(aabb a,aabb b){
		aabb c;
		c.pmin=make_float3(min(a.pmin.x,b.pmin.x),min(a.pmin.y,b.pmin.y),min(a.pmin.z,b.pmin.z));
		c.pmax=make_float3(max(a.pmax.x,b.pmax.x),max(a.pmax.y,b.pmax.y),max(a.pmax.z,b.pmax.z));
		return c;
	}
	float SurfaceArea(){
		float3 d=pmax-pmin;
		return 2*(d.x*d.y+d.x*d.z+d.y*d.z);
	}
	int MaxDim(){
		float3 d=pmax-pmin;
		if(d.x>d.y&&d.x>d.z) return 0;
		else if(d.y>d.z) return 1;
		else return 2;
	}
	float3 Centroid(){
		return (pmin+pmax)*0.5;
	}
};

struct wooptri
{
	aabb bounds;
	int id;
	float4 v0,v1,v2;
};

void woopifyTri(
	float3 v0,float3 v1,float3 v2,
	float4& woopv0,float4& woopv1,float4& woopv2
){
	float4x4 mtx;
	mtx.setCol(0,make_float4(v0-v2,0));
	mtx.setCol(1,make_float4(v1-v2,0));
	mtx.setCol(2,make_float4(cross(v0-v2,v1-v2),0));
	mtx.setCol(3,make_float4(v2,1));
	
	mtx.invert();
	
	woopv0=mtx.getRow(2);
	woopv1=mtx.getRow(0);
	woopv2=mtx.getRow(1);
}

void initTri(rtAccelerationStructureDesc &Desc,vector<wooptri>& tris)
{
	rtGeometryDesc* p=Desc.Inputs.pGeometryDescs;
	int nGeo=Desc.Inputs.NumDescs;
	vector<int> idxbuf;
	vector<float3> vtxbuf;
	
	tris.clear();
	for(int i=0;i<nGeo;i++){
		int idxcnt=p[i].IndexCount;
		int vtxcnt=p[i].VertexCount;
		idxbuf.resize(idxcnt);
		vtxbuf.resize(vtxcnt);
		rtCopyFromBuffer(p[i].IndexBuffer,idxbuf.data(),sizeof(int)*idxcnt);
		rtCopyFromBuffer(p[i].VertexBuffer,vtxbuf.data(),sizeof(float3)*vtxcnt);
		
		for(int j=0;j+2<idxcnt;j+=3){
			//assert(idxbuf[j]<vtxcnt&&idxbuf[j+1]<vtxcnt&&idxbuf[j+2]<vtxcnt);
			float3 v0=vtxbuf[idxbuf[j]];
			float3 v1=vtxbuf[idxbuf[j+1]];
			float3 v2=vtxbuf[idxbuf[j+2]];
			wooptri tri;
			tri.bounds=aabb(v0,v1,v2);
			tri.id=tris.size();
			woopifyTri(v0,v1,v2,tri.v0,tri.v1,tri.v2);
			tris.push_back(tri);
		}
	}
}

void buildBVH(vector<wooptri>& tris,vector<float4>& nodedata,vector<float4>& tridata)
{
	if (tris.size() <= 4) {
		aabb c0, c1;
		for (int i = 0; i < tris.size(); i++) {
			c0 = merge(c0, tris[i].bounds);
		}
		nodedata.push_back(make_float4(c0.pmin.x, c0.pmax.x, c0.pmin.y, c0.pmax.y));
		nodedata.push_back(make_float4(c1.pmin.x, c1.pmax.x, c1.pmin.y, c1.pmax.y));
		nodedata.push_back(make_float4(c0.pmin.z, c0.pmax.z, c1.pmin.z, c1.pmax.z));
		nodedata.push_back(make_float4(0, 0, 0, 0));
		int nodeidx = ~0;
		nodedata[3].x = *(float*)(&nodeidx);
		nodedata[3].y = *(float*)(&nodeidx);
		for (int i = 0; i < tris.size(); i++) {
			tridata.push_back(tris[i].v0);
			tridata.push_back(tris[i].v1);
			tridata.push_back(tris[i].v2);
		}
		uint terminator = 0x80000000;
		tridata.push_back(make_float4(*(float*)(&terminator), 0, 0, 0));
		return;
	}

	struct bucketinfo{
		int cnt;
		aabb bounds;
	};
	struct qnode{
		int l,r,dep;
		int fa,ci;
	};
	queue<qnode> q;
	const int nbuckets=12;
	bucketinfo buckets[nbuckets];
	
	q.push(qnode{0,(int)tris.size(),0,-1,0});
	while(!q.empty()){
		
		qnode u=q.front(); q.pop();
		//assert(u.dep<64);
		
		if(u.r-u.l>4){
			aabb cb;
			for(int i=u.l;i<u.r;i++){
				cb=merge(cb,tris[i].bounds.Centroid());
			}
			int dim=cb.MaxDim();
			
			for(int i=0;i<nbuckets;i++){
				buckets[i].cnt=0,buckets[i].bounds=aabb();
			}
			for(int i=u.l;i<u.r;i++){
				float3 c = tris[i].bounds.Centroid();
				float l=(&cb.pmin.x)[dim];
				float r=(&cb.pmax.x)[dim];
				float x=(&c.x)[dim];
				int b=nbuckets*(x-l)/(r-l);
				if(b==nbuckets) b--;
				buckets[b].cnt++;
				buckets[b].bounds=merge(buckets[b].bounds,tris[i].bounds);
			}
			
			float mincost=1e18;
			int mincostidx=0;
			aabb c0,c1;
			
			for(int i=0;i<nbuckets;i++){
				aabb b0,b1;
				int cnt0=0,cnt1=0;
				for(int j=0;j<=i;j++){
					b0=merge(b0,buckets[j].bounds);
					cnt0+=buckets[j].cnt;
				}
				for(int j=i+1;j<nbuckets;j++){
					b1=merge(b1,buckets[j].bounds);
					cnt1+=buckets[j].cnt;
				}
				float cost=cnt0*b0.SurfaceArea()+cnt1*b1.SurfaceArea();
				if(cost<mincost){
					mincost=cost;
					mincostidx=i;
					c0=b0,c1=b1;
				}
			}
			wooptri *pmid=std::partition(&tris[u.l],&tris[u.r-1]+1,
				[=](wooptri &pi){
					float3 c = pi.bounds.Centroid();
					float l=(&cb.pmin.x)[dim];
					float r=(&cb.pmax.x)[dim];
					float x=(&c.x)[dim];
					int b=nbuckets*(x-l)/(r-l);
					if(b==nbuckets) b--;
					return b<=mincostidx;
				});
			int mid=pmid-&tris[u.l];
			
			int nodeidx=nodedata.size();
			if(u.fa>=0){
				if(u.ci==0) nodedata[u.fa+3].x=*(float*)(&nodeidx);
				else nodedata[u.fa+3].y=*(float*)(&nodeidx);
			}
			nodedata.push_back(make_float4(c0.pmin.x,c0.pmax.x,c0.pmin.y,c0.pmax.y));
			nodedata.push_back(make_float4(c1.pmin.x,c1.pmax.x,c1.pmin.y,c1.pmax.y));
			nodedata.push_back(make_float4(c0.pmin.z,c0.pmax.z,c1.pmin.z,c1.pmax.z));
			nodedata.push_back(make_float4(0,0,0,0));
			
			q.push({u.l,mid,u.dep+1,nodeidx,0});
			q.push({mid,u.r,u.dep+1,nodeidx,1});
		}
		else{
			int nodeidx=tridata.size();
			nodeidx=~nodeidx;
			if(u.fa>=0){
				if(u.ci==0) nodedata[u.fa+3].x=*(float*)(&nodeidx);
				else nodedata[u.fa+3].y=*(float*)(&nodeidx);
			}
			for(int i=u.l;i<u.r;i++){
				tridata.push_back(tris[i].v0);
				tridata.push_back(tris[i].v1);
				tridata.push_back(tris[i].v2);
			}
			uint terminator=0x80000000;
			tridata.push_back(make_float4(*(float*)(&terminator),0,0,0));
		}
		
	}
}

extern "C"
void rtBuildAccelerationStructure(rtAccelerationStructureDesc Desc)
{
	vector<wooptri> tris;
	initTri(Desc,tris);
	
	vector<float4> nodedata;
	vector<float4> tridata;
	
	buildBVH(tris,nodedata,tridata);
	
}
