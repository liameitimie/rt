#include <hip/hip_runtime.h>
#include "rt_types.h"
#include "cudaChecker.h"

extern "C"
rtBuffer rtCreateBuffer(uint Size, uint Stride = 0)
{
	void* ptr;
	check(hipMalloc(&ptr, Size));
	rtBuffer res;
	res.StartAddress = (uint64)ptr;
	res.Size = Size;
	res.Stride = Stride;
	return res;
}

extern "C"
void rtReleaseBuffer(rtBuffer Buffer)
{
	check(hipFree((void*)Buffer.StartAddress));
}

extern "C"
void rtCopyToBuffer(rtBuffer Buffer, void* ptr, uint Size)
{
	check(hipMemcpy((void*)Buffer.StartAddress, ptr, Size, hipMemcpyHostToDevice));
}

extern "C"
void rtCopyFromBuffer(rtBuffer Buffer, void* ptr, uint Size)
{
	check(hipMemcpy(ptr, (void*)Buffer.StartAddress, Size, hipMemcpyDeviceToHost));
}
