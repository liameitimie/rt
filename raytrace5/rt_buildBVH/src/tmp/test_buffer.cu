#include "hip/hip_runtime.h"
#include <iostream>
#include "rtBuffer.h"

using namespace std;

int n=5;
int a[]={3,43,1,5,6};
int b[]={4,5,11,23,1};

__global__
void kernel(rtBufferView<int> a,rtBufferView<int> b,rtBufferView<int> c,int n){
	int idx=threadIdx.x;
	if(idx<n) c[idx]=a[idx]+b[idx];
}

int main(){
	int size=n*sizeof(int);
	rtBuffer buf1=rtCreateBuffer(size);
	rtBuffer buf2=rtCreateBuffer(size);
	rtBuffer buf3=rtCreateBuffer(size);
	
	rtCopyToBuffer(buf1,a,size);
	rtCopyToBuffer(buf2,b,size);
	
	kernel<<<1,n>>>(buf1,buf2,buf3,n);
	check_kernel();
	
	rtCopyFromBuffer(buf3,a,size);
	
	for(int i=0;i<n;i++){
		cout<<a[i]<<endl;
	}
	
	return 0;
}
