#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "rt_types.h"
#include "cudaChecker.h"

template<class T>
struct rtBufferView
{
	uint64 StartAddress;
	uint32 Size;
	uint32 Stride;
	
	rtBufferView(rtBuffer Buffer)
	{
		StartAddress = Buffer.StartAddress;
		Size = Buffer.Size;
		Stride = Buffer.Stride;
	}
	
	__device__ T& operator[](int i)
	{
		return *(T*)(StartAddress+i*(Stride?Stride:sizeof(T)));
	}
};

__global__
void kernel(rtBufferView<int> a,rtBufferView<int> b,rtBufferView<int> c,int n){
	int idx=threadIdx.x;
	if(idx<n) c[idx]=a[idx]+b[idx];
}

extern "C"
void add(rtBuffer buf1,rtBuffer buf2,rtBuffer buf3,int n)
{
	kernel<<<1,n>>>(buf1,buf2,buf3,n);
	check_kernel();
}

