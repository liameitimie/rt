#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

using uint64=unsigned long long;
using func_ptr=int(*)(int,int); 

#define check(call)\
{\
    const hipError_t error=call;\
    if(error!=hipSuccess)\
    {\
        printf("ERROR: %s:%d,",__FILE__,__LINE__);\
        printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
        exit(1);\
    }\
}
#define check_kernel()\
{\
	check(hipDeviceSynchronize());\
	const hipError_t error=hipGetLastError();\
	if(error!=hipSuccess)\
    {\
        printf("ERROR: %s:%d,",__FILE__,__LINE__);\
        printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
        exit(1);\
    }\
}

__global__
void kernel(uint64 p,int a,int b,int* c){
	func_ptr f=(func_ptr)p;
	*c=f(a,b);
}

__device__
int f(int a,int b){
	return a+b;
}

__device__
uint64 f_ptr=(uint64)f;

int main(){
	uint64 f_h;
	check(hipMemcpyFromSymbol(&f_h,HIP_SYMBOL(f_ptr),sizeof(f_h)));
	
	int* c_d;
	check(hipMalloc(&c_d,sizeof(int)));
	
	kernel<<<1,1>>>(f_h,2,3,c_d);
	check_kernel();
	
	int c;
	check(hipMemcpy(&c,c_d,sizeof(c),hipMemcpyDeviceToHost));
	
	cout<<c<<endl;
	
	return 0;
}
