#include <algorithm>
#include <mesh.h>
#include <hip/hip_runtime.h>

using namespace std;

void Mesh::build_mesh_data(vector<Vertex>& vertices,vector<int>& indices,MeshData& mesh_data){
    void* vbuffer;
    void* ibuffer;

    uint64 vbuffer_sz=sizeof(Vertex)*vertices.size();
    uint64 ibuffer_sz=sizeof(int)*indices.size();
    hipMalloc(&vbuffer,vbuffer_sz);
    hipMalloc(&ibuffer,ibuffer_sz);
    hipMemcpy(vbuffer,vertices.data(),vbuffer_sz,hipMemcpyHostToDevice);
    hipMemcpy(ibuffer,indices.data(),ibuffer_sz,hipMemcpyHostToDevice);

    mesh_data={
        span<Vertex>(vertices),
        span<int>(indices),
        (uint64)vbuffer,
        (uint64)ibuffer,
    };
}